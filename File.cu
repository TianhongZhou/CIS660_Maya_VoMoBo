#include "hip/hip_runtime.h"
#include "Header.h"
#include <hip/hip_runtime.h>
#include <>
#include <cmath>

constexpr double EPSILON = 1e-6;

__device__ bool insideTriangleYZ(
    double y0, double z0,
    double y1, double z1,
    double y2, double z2,
    double y, double z)
{
    double w0 = (y - y0) * (z1 - z0) - (z - z0) * (y1 - y0);
    double w1 = (y - y1) * (z2 - z1) - (z - z1) * (y2 - y1);
    double w2 = (y - y2) * (z0 - z2) - (z - z2) * (y0 - y2);
    return (w0 >= 0 && w1 >= 0 && w2 >= 0) || (w0 <= 0 && w1 <= 0 && w2 <= 0);
}

__device__ bool intersectTriangleX(
    double x0, double y0, double z0,
    double x1, double y1, double z1,
    double x2, double y2, double z2,
    double y, double z,
    double& xi)
{
    double A = (y1 - y0) * (z2 - z0) - (z1 - z0) * (y2 - y0);
    if (fabs(A) < EPSILON) return false;
    double B = (z1 - z0) * (x2 - x0) - (x1 - x0) * (z2 - z0);
    double C = (x1 - x0) * (y2 - y0) - (y1 - y0) * (x2 - x0);
    double D = -(A * x0 + B * y0 + C * z0);
    xi = (-D - B * y - C * z) / A;
    return true;
}

__global__ void voxelizeKernel(
    unsigned char* G, int res,
    double minX, double maxX,
    double minY, double maxY,
    double minZ, double maxZ,
    int numTris,
    const double* v0x, const double* v0y, const double* v0z,
    const double* v1x, const double* v1y, const double* v1z,
    const double* v2x, const double* v2y, const double* v2z)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x >= res || y >= res || z >= res) return;

    double fx = (maxX - minX) / (res - 1.0);
    double fy = (maxY - minY) / (res - 1.0);
    double fz = (maxZ - minZ) / (res - 1.0);
    double vx = minX + x * fx + fx * 0.5;
    double vy = minY + y * fy + fy * 0.5;
    double vz = minZ + z * fz + fz * 0.5;

    int cnt = 0;
    for (int t = 0; t < numTris; ++t) {
        if (!insideTriangleYZ(v0y[t], v0z[t], v1y[t], v1z[t], v2y[t], v2z[t], vy, vz)) continue;
        double xi;
        if (!intersectTriangleX(v0x[t], v0y[t], v0z[t], v1x[t], v1y[t], v1z[t], v2x[t], v2y[t], v2z[t], vy, vz, xi)) continue;
        if (vx >= xi) ++cnt;
    }
    size_t idx = (x * res + y) * res + z;
    G[idx] = (cnt & 1) ? 1 : 0;
}

extern "C" void voxelizeOnGpu(
    const double* v0x, const double* v0y, const double* v0z,
    const double* v1x, const double* v1y, const double* v1z,
    const double* v2x, const double* v2y, const double* v2z,
    int numTris, int res,
    double minX, double maxX,
    double minY, double maxY,
    double minZ, double maxZ,
    unsigned char* outG)
{
    size_t triB = numTris * sizeof(double);
    size_t voxB = size_t(res) * res * res * sizeof(unsigned char);

    double* d0x, * d0y, * d0z, * d1x, * d1y, * d1z, * d2x, * d2y, * d2z;
    hipMalloc(&d0x, triB); hipMalloc(&d0y, triB); hipMalloc(&d0z, triB);
    hipMalloc(&d1x, triB); hipMalloc(&d1y, triB); hipMalloc(&d1z, triB);
    hipMalloc(&d2x, triB); hipMalloc(&d2y, triB); hipMalloc(&d2z, triB);
    hipMemcpy(d0x, v0x, triB, hipMemcpyHostToDevice);
    hipMemcpy(d0y, v0y, triB, hipMemcpyHostToDevice);
    hipMemcpy(d0z, v0z, triB, hipMemcpyHostToDevice);
    hipMemcpy(d1x, v1x, triB, hipMemcpyHostToDevice);
    hipMemcpy(d1y, v1y, triB, hipMemcpyHostToDevice);
    hipMemcpy(d1z, v1z, triB, hipMemcpyHostToDevice);
    hipMemcpy(d2x, v2x, triB, hipMemcpyHostToDevice);
    hipMemcpy(d2y, v2y, triB, hipMemcpyHostToDevice);
    hipMemcpy(d2z, v2z, triB, hipMemcpyHostToDevice);

    unsigned char* dG;
    hipMalloc(&dG, voxB);

    dim3 blk(8, 8, 8), grd((res + 7) / 8, (res + 7) / 8, (res + 7) / 8);
    voxelizeKernel << <grd, blk >> > (dG, res, minX, maxX, minY, maxY, minZ, maxZ, numTris,
        d0x, d0y, d0z, d1x, d1y, d1z, d2x, d2y, d2z);
    hipDeviceSynchronize();

    hipMemcpy(outG, dG, voxB, hipMemcpyDeviceToHost);

    hipFree(dG);
    hipFree(d0x); hipFree(d0y); hipFree(d0z);
    hipFree(d1x); hipFree(d1y); hipFree(d1z);
    hipFree(d2x); hipFree(d2y); hipFree(d2z);
}